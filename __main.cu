#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"


#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define nx 200
#define ny 100


__global__ void add(float a, float b) {}

void draw_image(int* data, uint32_t length)
{
    fprintf(stderr, "Entering draw_image func\n");
    /*
    for (uint32_t i = 0; i < length; i++)
    {
        r = rand() % 255;
        g = rand() % 255;
        b = rand() % 255;
        data[i] = (0xff << 24) | (r << 16) | (g << 8) | b;
        //printf("RGB = (%02x) (%02x) (%02x) data=%08x\n", r, g, b, data[i]);
    }
    */

    //int stbi_write_png(char const *filename, int w, int h, int comp, const void *data, int stride_in_bytes);
    int val = stbi_write_png("someshit.png", nx, ny, 4, data, 0);

    if (!val) {
        fprintf(stderr, "SOME SHIT FAILED\n");
    } else {
        fprintf(stderr, "stb_write returned %d\n", val);
    }

    //int val = stbi_write_bmp("someshit.bmp", WIDTH, HEIGHT, 3, data);
    val = stbi_write_tga("someshit.tga", nx, ny, 4, data);

    if (!val) {
        fprintf(stderr, "SOME SHIT FAILED\n");
    } else {
        fprintf(stderr, "stb_write returned %d\n", val);
    }
}

void draw_some_pixels(int* data)
{
    fprintf(stderr, "Entering draw_some_pixels() func\n");

    for (int j = ny - 1; j >= 0; j--)
    {
        for (int i = 0; i < nx; i++)
        {
            float r = (float)i / (float)nx;
            float g = (float)j / (float)ny;
            float b = 0.2f;

            int ir = 255.99 * (float)r;
            int ig = 255.99 * (float)g;
            int ib = 255.99 * (float)b;
            int index = (j * nx) + i;
            //fprintf(stderr, "[(i)=%d, (j)=%d   Index = %03d]   ", j, i, index);
            data[index] = (0xff << 24) | (ib << 16) | (ig << 8) | ir;
        }
        add<<<1,1>>>(3.0f, 3.0f);
        //fprintf(stderr, "\n");
    }

    fprintf(stderr, "Leaving draw_some_pixels()\n");
}


int main()
{
    setvbuf(stdout, 0, _IOLBF, 4096);
    int c = int(43);
    int* data;
    data = (int*)malloc(sizeof(uint32_t) * nx * ny);

    if ( data == NULL ) 
    {
        fprintf(stderr, "Some tragic shit happend while allocating memory\n");
        return EXIT_FAILURE;
    }

    //stbi_flip_vertically_on_write(-1); // flag is non-zero to flip data vertically

    draw_some_pixels(data);
    draw_image(data, nx * ny);

    free(data);

    return EXIT_SUCCESS;
}
